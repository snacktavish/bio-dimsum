#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#define numRand 10000

#define __DIMSUM_STD_R 6371.0f
#define __DIMSUM_mode_northeast 0
#define __DIMSUM_mode_southeast 1
#define __DIMSUM_mode_northwest 2
#define __DIMSUM_mode_southwest 3
#define __DIMSUM_NODE_lat 0
#define __DIMSUM_NODE_lon 1
#define __DIMSUM_PARAMI_RANDINDEX 0
#define __DIMSUM_PARAMI_RANDMAX 1
#define __DIMSUM_PARAMI_SB_INDEX 2
#define __DIMSUM_PARAMI_HB_INDEX 3
#define __DIMSUM_PARAMI_GENERATION 4
#define __DIMSUM_PARAMI_NUMCHILDREN 5
#define __DIMSUM_XYFUNCTION_xsize 0
#define __DIMSUM_XYFUNCTION_ysize 1
#define __DIMSUM_XYFUNCTION_meta_length 4
#define __DIMSUM_XYFUNCTION_Fdim 3
#define PI 3.1415926536
#define __DIMSUM_MIN_D 0.000001
#define __DIMSUM_KA2_EPSILON 0.00000000001745329252
#define __DIMSUM_EPSILON 0.00000017453292520000

//double* randArray;
float* sb_DATA;
int* sb_META;
//int* sb_SIZE;
float* hb_DATA;
int* hb_META;
//int* hb_SIZE;

texture<float, 1, hipReadModeElementType> randArray;
texture<float, 3, hipReadModeElementType> softborderDATA;
texture<int, 2, hipReadModeElementType> softborderMETA;
texture<float, 1, hipReadModeElementType> softborderSIZE;
texture<float, 3, hipReadModeElementType> hardborderDATA;
texture<int, 2, hipReadModeElementType> hardborderMETA;
texture<float, 1, hipReadModeElementType> hardborderSIZE;


/*
void setRandArray(double* _randArray) {
	randArray = new double[numRand];
	for(int i=0;i<numRand;i++)
		randArray[i] = _randArray[i];

}*/
/*
void setArrays(float* _sb_DATA,int* _sb_META,int* _sb_SIZE,float* _hb_DATA,int* _hb_META,int* _hb_SIZE) {
	sb_SIZE = new int[__DIMSUM_XYFUNCTION_Fdim];
	hb_SIZE = new int[__DIMSUM_XYFUNCTION_Fdim];
	for(int i=0;i<__DIMSUM_XYFUNCTION_Fdim;i++) {
		sb_SIZE[i] = _sb_SIZE[i];
		hb_SIZE[i] = _hb_SIZE[i];
	}

	int sb_length = sb_SIZE[0];
	int hb_length = hb_SIZE[0];
	sb_META = new int[__DIMSUM_XYFUNCTION_meta_length * sb_length];
	hb_META = new int[hb_length * __DIMSUM_XYFUNCTION_meta_length];
	for(int i=0;i<(__DIMSUM_XYFUNCTION_meta_length * sb_length);i++) {
		sb_META[i] = _sb_META[i];
	}

	for(int i=0;i<(hb_length * __DIMSUM_XYFUNCTION_meta_length);i++) {
			hb_META[i] = _hb_META[i];
		}
	sb_DATA = new float[sb_length * sb_SIZE[1] * sb_SIZE[2]];
	hb_DATA = new float[hb_length * hb_SIZE[1] * hb_SIZE[2]];
	for(int i=0;i<(sb_length * sb_SIZE[1] * sb_SIZE[2]);i++) {
		sb_DATA[i] = _sb_DATA[i];
		}
	for(int i=0;i<(hb_length * hb_SIZE[1] * hb_SIZE[2]);i++) {
			hb_DATA[i] = _hb_DATA[i];
			}
}*/
extern "C" {
__device__
double toDeg(double data) { return data*180.0f/PI;}

__device__
double toRad(double data) { return data*PI/180.0f;}
/*
__device__
double abs(double x) {
	if (x < 0)
		return -x;
	return x;
}

__device__
double min(double x, double y) {
	if (x < y)
		return x;
	return y;
}
*/
__device__
double mod(double y, double x)
{
	return (y - (x*floor(y/x)));
}

__device__
double geq (double lat1, double  sb_lat_bt) {
	if(lat1 >= sb_lat_bt)
		return 1;
	else //if (lat1 < sb_lat_bt)
		return -1;

}

__device__
double nextRand(int* _index) {
	/*if(_index[__DIMSUM_PARAMI_RANDINDEX]>=_index[__DIMSUM_PARAMI_RANDMAX]) {
		System.err.println("nextRand(): index>numRand");
		System.exit(-1);
	}*/
	double r =  tex1D(randArray,atomicAdd(&(_index[__DIMSUM_PARAMI_RANDINDEX]),1));
	//_index[__DIMSUM_PARAMI_RANDINDEX]++;

	return r;
}
/*
__device__
int getMetaIndex_sb(int i, int type) {
	return i+type*tex1D(softborderSIZE,0);
}

__device__
int getMetaIndex_hb(int i, int type) {
	return i+type*tex1D(hardborderSIZE,0);
}
*/

__device__
float sb_f(int i, int y,int x) {
	return tex3D(softborderDATA,i,x,y);
}


__device__
float hb_f(int i, int y,int x) {
	return tex3D(hardborderDATA,i,x,y);
}


__device__
int sb_toX(int i,double lon, double minlon, double maxlon, int _size) {
	if( lon >= maxlon )
		return tex2D(softborderMETA,i, _size) -1;
	if( lon <= minlon )
		return 0;
	return (int)(tex2D(softborderMETA,i, _size)  * (lon-minlon)/(maxlon-minlon));
}

__device__
int hb_toX(int i,double lon, double minlon, double maxlon, int _size) {
	if( lon >= maxlon )
		return tex2D(hardborderMETA,i, _size) -1;
	if( lon <= minlon )
		return 0;
	return (int)(tex2D(hardborderMETA,i, _size)  * (lon-minlon)/(maxlon-minlon));
}

__device__
int getMode(double crs) {
	int mode = -1;
	if( crs >= 0 && crs <= PI/2 ) {		// JMB COMMENT -- DETERMINES IF COURSE IS NORTHEAST...OR IS THIS SOUTHEAST, SINCE INPUT MAPS ARE FLIPPED??
		mode = __DIMSUM_mode_northeast;
	}
	else if( crs > PI/2 && crs <= PI ) {			// JMB COMMENT -- Determines if course is southeast.  Note that this coordinate system is upside down relative to the standard.
		mode = __DIMSUM_mode_southeast;
	}
	else if( crs > PI && crs <= 3*PI/2 ) {  // JMB COMMENT -- DETERMINES IF COURSE IS SOUTHWEST
		mode = __DIMSUM_mode_southwest;
	}
	else if( crs > 3*PI/2 && crs <= 2*PI ) {   // JMB COMMENT -- 10.19.09 -- DETERMINES IF COURSE IS NORTHWEST
		mode = __DIMSUM_mode_northwest;
	}
	return mode;
}

__device__
int northORsouth(int i) {
	if(i == __DIMSUM_mode_northeast || i == __DIMSUM_mode_northwest)
		return 1;
	else return -1;
}

__device__
int eastORwest(int i) {
	if(i == __DIMSUM_mode_northeast || i == __DIMSUM_mode_southeast)
		return 1;
	else return -1;
}

__device__
double getI(double lat1, double minlat, double latspace, int dir) {
	double ilat;
	double mlat = lat1-minlat ;//toRad(minlat2);
	if(dir == 1)
		ilat = minlat + (latspace*floor(mlat/latspace)) + latspace;
	else
		ilat = minlat + (latspace*floor(mlat/latspace));

	return ilat;
}


__device__
double dfromll(double lats, double lons, double latf, double lonf)			// JMB COMMENT -- 10.19.09 -- DETERMINES DISTANCE (D) AND COURSE (CRS) FROM ENDING LAT/LON
{
	double dlat = latf - lats;
	double dlon = lonf - lons;
	double a = pow(sin(dlat/2),2) + cos(lats)*cos(latf)*pow(sin(dlon/2),2);

	return __DIMSUM_STD_R * 2 * atan2(sqrt(a),sqrt(1-a));
}


/*private static double crsfromll( double lats,double lons, double latf,  double lonf)			// JMB COMMENT -- 10.19.09 -- DETERMINES DISTANCE (D) AND COURSE (CRS) FROM ENDING LAT/LON
{
	double dlon = lonf - lons;
	return mod(atan2(sin(dlon)*cos(latf),cos(lats)*sin(latf)-sin(lats)*cos(latf)*cos(dlon)),2*PI);
}*/

/*
void llffromdcrs(double lld_lats,double lld_lons, double lld_latf,  double lld_lonf, double lld_d, double lld_crs)			// JMB COMMENT -- 10.19.09 -- DETERMINES ENDING LAT/LON FROM DISTANCE (D) AND COURSE (CRS)
{
	lld_latf = asin(sin(lld_lats)*cos(lld_d/__DIMSUM_STD_R)+cos(lld_lats)*sin(lld_d/__DIMSUM_STD_R)*cos(lld_crs));
	lld_lonf = lld_lons + atan2(sin(lld_crs)*sin(lld_d/__DIMSUM_STD_R)*cos(lld_lats),cos(lld_d/__DIMSUM_STD_R)-sin(lld_lats)*sin(lld_latf));
}*/


__device__
double latdfromlon3(double lat1,double lon1, double lat2,  double lon2,  double lon3) // // Implementation by JMB from http://williams.best.vwh.net/avform.htm#Par  -- lat=atan(  (sin(lat1)*cosf(lat2)*sin(lon-lon2)-sin(lat2)*cosf(lat1)*sin(lon-lon1))  /  (cosf(lat1)*cosf(lat2)*sin(lon1-lon2))  )
{
	//double lon3 = toRad(lon3o);//*PI/180;
	double latf = atan(((sin(lat1)*cos(lat2)*sin(lon3-lon2))-(sin(lat2)*cos(lat1)*sin(lon3-lon1)))/(cos(lat1)*cos(lat2)*sin(lon1-lon2)));

	return dfromll(lat1,lon1,latf, lon3);
}


__device__
double londfromlat3(double lat1,double lon1, double lat2,  double lon2, double lat3) // in degrees		JMB COMMENT -- 10.19.09 -- FINDS LONGITUDE AT WHICH NEAREST LAT IS CROSSED W/
{														//								GREAT CIRCLE DISTANCE
	// from http://williams.best.vwh.net/avform.htm#Par
	//double lat3 = toRad(lat3o);// * PI / 180.0f;
	double d;

	double A = sin(lat1)*cos(lat2)*cos(lat3)*sin(lon1-lon2);
	double B = sin(lat1)*cos(lat2)*cos(lat3)*cos(lon1-lon2) - cos(lat1)*sin(lat2)*cos(lat3);
	double C = cos(lat1)*cos(lat2)*sin(lat3)*sin(lon1-lon2);
	double lon = atan2(B,A);
	if (abs(C) >sqrt(pow(A,2) + pow(B,2))) {
		d = 10000000;
	} else if( lat1 == lat3 ) {
		d = 0;
	}
	else {
		double dlon = acos(C/sqrt(pow(A,2)+pow(B,2)));
		double lon3_1=(mod((lon1+dlon+lon+PI),(2*PI))-PI);
		double lon3_2=(mod((lon1-dlon+lon+PI),(2*PI))-PI);
		double lon3_1_d_D = dfromll(lat1, lon1,lat3,lon3_1);
		double lon3_2_d_D = dfromll(lat1, lon1,lat3, lon3_2);

		if( ((lon3_1 >= lon1 && lon3_1 <= lon2) || (lon3_1 <= lon1 && lon3_1 >= lon2)) && (lon3_1_d_D < lon3_2_d_D ))
			d = lon3_1_d_D;
		else
			d = lon3_2_d_D;

	}

	return d;
}

__device__
double setX(double X, double x_bt, double prefix, double dXY){
	double r = X;
	if (((prefix > 0 && r < x_bt) || (prefix < 0 && r > x_bt)) && dXY == 0)
	    r = x_bt + __DIMSUM_EPSILON*prefix;
	return r;
}


__device__
double ka2(double lat1,double minlat, double sb_latspace, int dirLat,int* _index) {
	double ilat = getI(lat1,minlat,sb_latspace, dirLat);
	if (lat1 == ilat)	// Checking to see if individual is starting RIGHT ON a lat line.
	{					// If so, pushes it barely off in a random direction.
		if (nextRand(_index) > 0.5)
			lat1 += __DIMSUM_KA2_EPSILON;
		else
			lat1 -= __DIMSUM_KA2_EPSILON;
		ilat = getI(lat1,minlat,sb_latspace, dirLat);
	}
	return ilat;
}

__device__
void migrate(double* node, double* dA, int* rm,int* parami, double* paramv, int id) {
	double d = dA[id];
	int sb = parami[__DIMSUM_PARAMI_SB_INDEX];
	int hb = parami[__DIMSUM_PARAMI_HB_INDEX];
	double lat1 = toRad(node[id*2+__DIMSUM_NODE_lat]);
	double lon1 = toRad(node[id*2+__DIMSUM_NODE_lon]);
	double minlat = toRad(paramv[0]);
	double maxlat = toRad(paramv[1]);
	double minlon = toRad(paramv[2]);
	double maxlon = toRad(paramv[3]);
	double sb_lonspace = toRad(paramv[4]);
	double sb_latspace = toRad(paramv[5]);
	double hb_lonspace = toRad(paramv[6]);
	double hb_latspace = toRad(paramv[7]);


	//END DECIMAL DEGREE

	double step_d = 0.0001;
	double sb_lat_bt,sb_lon_bt,hb_lat_bt,hb_lon_bt,hb_dd,sb_dd;				// JMB -- Using this to keep track of lat/lon value for border reflections and adjusting inexact positions, if necessary
	double crs = nextRand(parami) * 2 * PI;	// Modified by JMB -- 4.5.10
	double lld_d, lld_latf=0, lld_lonf=0,lld_crs;

	while( d >= __DIMSUM_MIN_D ) {
		//System.out.println(lat1+ " "+lon1+" "+d);
		lld_d = d;					// JMB COMMENT -- SETS D INTERNALLY IN LLD OBJECT
		//lld_r = 6371;				// radius of spherical earth in km (TODO: use value(# or units?) from xml)
		//lld_lats = lat1;//toRad(lat1);// / 180.0f * PI; //setLatSDDeg(lld,lat1);		// JMB COMMENT -- CONVERTS LATITUDE FROM DECIMAL DEGREES TO RADIANS
		//lld_lons = lon1;//toRad(lon1);// / 180.0f * PI;//	setLonSDDeg(lld,lon1);		// JMB COMMENT -- CONVERTS LONGITUDE FROM DECIMAL DEGREES TO RADIANS
		lld_crs = crs;				// JMB COMMENT -- SETS CRS INTERNALLY WITHIN LLD OBJECT
		lld_latf = asin(sin(lat1)*cos(lld_d/__DIMSUM_STD_R)+cos(lat1)*sin(lld_d/__DIMSUM_STD_R)*cos(lld_crs));// JMB COMMENT -- GETS ENDING LAT/LON FROM DISTANCE AND COURSE AND STORES INTERNALLY IN LLD OBJECT
		lld_lonf = lon1 + atan2(sin(lld_crs)*sin(lld_d/__DIMSUM_STD_R)*cos(lat1),cos(lld_d/__DIMSUM_STD_R)-sin(lat1)*sin(lld_latf));

		hb_dd=100000000;
		sb_dd=100000000;
		int hb_dx=0,hb_dy=0,sb_dx=0,sb_dy=0;

		int mode = getMode(crs);

		sb_lat_bt = ka2(lat1,minlat,sb_latspace,northORsouth(mode),parami);
		double i1_d = londfromlat3(lat1,lon1,lld_latf,lld_lonf,sb_lat_bt);		// JMB COMMENT -- FINDS COORDINATES FOR NEAREST LAT BORDER CROSSING

		sb_lon_bt = ka2(lon1,minlon,sb_lonspace,eastORwest(mode),parami);
		double i2_d = latdfromlon3(lat1,lon1,lld_latf,lld_lonf,sb_lon_bt);		// JMB COMMENT -- FINDS COORDINATES FOR NEAREST LON BORDER CROSSING

		if( i1_d <= i2_d  && i1_d < d) {
			sb_dd = i1_d;
			sb_dy = northORsouth(mode);
		}
		else if( i2_d < d ) {
			sb_dd = i2_d;
			sb_dx = eastORwest(mode);
		}

		hb_lat_bt = ka2(lat1,minlat,hb_latspace,northORsouth(mode),parami);
		i1_d = londfromlat3(lat1,lon1,lld_latf,lld_lonf,hb_lat_bt);		// JMB COMMENT -- FINDS COORDINATES FOR NEAREST LAT BORDER CROSSING

		hb_lon_bt = ka2(lon1,minlon,hb_lonspace,eastORwest(mode),parami);
		i2_d = latdfromlon3(lat1,lon1,lld_latf,lld_lonf,hb_lon_bt);		// JMB COMMENT -- FINDS COORDINATES FOR NEAREST LON BORDER CROSSING

		if( i1_d <= i2_d  && i1_d < d) {
			hb_dd = i1_d;
			hb_dy = northORsouth(mode);
		}
		else if( i2_d < d ) {
			hb_dd = i2_d;
			hb_dx = eastORwest(mode);
		}

		if( d < min(sb_dd,hb_dd) ) {	// JMB COMMENT -- 10.20.09 -- DISPERSAL OCCURS BEFORE PIXEL BOUNDARY IS CROSSED,
			lld_d = d;
			//*********************** Fudging to keep poorly estimated positions (due to step_d alterations) from crossing border boundaries inadvertently ***********************
			// IS THIS CHECK NECESSARY AT THIS POINT IN THE LOOP?  PERHAPS NOT, BUT SHOULD MAKE SURE BEFORE REMOVING IT.

	       // node[id*2+__DIMSUM_NODE_lat] = toDeg(lld_latf);//getLatFDDeg(lld);
	        //node[id*2+__DIMSUM_NODE_lon] = toDeg(lld_lonf);//getLonFDDeg(lld);
			lat1 = setX(lld_latf,sb_lat_bt,geq(lat1,sb_lat_bt),0);
			lon1 = setX(lld_lonf,sb_lon_bt,geq(lon1,sb_lon_bt),0);
			lat1 = setX(lld_latf,hb_lat_bt,geq(lat1,hb_lat_bt),0);
			lon1 = setX(lld_lonf,hb_lon_bt,geq(lon1,hb_lon_bt),0);
			d=0;
		}
		else if( abs(sb_dd-hb_dd) < step_d || sb_dd < hb_dd ) {	// JMB COMMENT -- 10.20.09 -- BOTH SOFT AND HARD PIXEL BOUNDARIES WILL BE CROSSED
			// both soft & hard must be checked at the same time -- but the order is up to you
			// I arbitrarily chose to check hard borders first
			if(abs(sb_dd-hb_dd) < step_d )
			if(  nextRand(parami) <= hb_f(hb,hb_toX(hb,lon1,minlon,maxlon,__DIMSUM_XYFUNCTION_xsize)+hb_dx,hb_toX(hb,lat1,minlat,maxlat,__DIMSUM_XYFUNCTION_ysize)+hb_dy) ) {	// JMB COMMENT -- FINDS HARD BORDER VALUE FOR NEXT PIXEL WITH RESPECT TO LONGITUDE AND CHECKS TO SEE IF INDIVIDUAL SURVIVES HARD BORDER CROSSING.
				rm[id] =  1; //continue childrenloop; // this exits the travel loop immediately, so the current child never gets added to the next generation
				return;
			}

			if( nextRand(parami) <= sb_f(sb,sb_toX(sb,lon1,minlon,maxlon,__DIMSUM_XYFUNCTION_xsize)+sb_dx, sb_toX(sb,lat1,minlat,maxlat,__DIMSUM_XYFUNCTION_ysize)+sb_dy) ) {
				// failed the soft border-- stop before border, reflect back, update d, and continue
				crs = nextRand(parami) * 2 * PI;// / 4+3*PI/4;
				sb_dy = 0;
				sb_dx = 0;
				lld_d = (sb_dd-step_d);
			} else {
				lld_d = sb_dd+step_d;
			}

			d-= lld_d;
			lld_latf = asin(sin(lat1)*cos(lld_d/__DIMSUM_STD_R)+cos(lat1)*sin(lld_d/__DIMSUM_STD_R)*cos(lld_crs));
			lld_lonf = lon1 + atan2(sin(lld_crs)*sin(lld_d/__DIMSUM_STD_R)*cos(lat1),cos(lld_d/__DIMSUM_STD_R)-sin(lat1)*sin(lld_latf));
			// Fudging to keep poorly estimated positions from crossing boundaries
			lat1 = setX(lld_latf,sb_lat_bt,geq(lat1,sb_lat_bt),sb_dy);
			lon1 = setX(lld_lonf,sb_lon_bt,geq(lon1,sb_lon_bt),sb_dx);
		}
		else {
			if( nextRand(parami) <= hb_f(hb,hb_toX(hb,lon1,minlon,maxlon,__DIMSUM_XYFUNCTION_xsize)+hb_dx,hb_toX(hb,lat1,minlat,maxlat,__DIMSUM_XYFUNCTION_ysize)+hb_dy) ) {
				rm[id] =  1; //continue childrenloop; // this exits the travel loop immediately, so the current child never gets added to the next generation
				return;						// JMB -- Would this lead to pruning problems?
			} else {
				lld_d = hb_dd+step_d;
				d-=lld_d;
				//TODO: check lat1 = toDeg(lld_latf);
				//TODO: check lon1 = toDeg(lld_lonf);
				lat1 = setX(lld_latf,hb_lat_bt,geq(lat1,hb_lat_bt),hb_dy);
				lon1 = setX(lld_lonf,hb_lon_bt,geq(lon1,hb_lon_bt),hb_dx);
			}
		}
	}
	node[id*2+__DIMSUM_NODE_lat] = toDeg(lat1);
	node[id*2+__DIMSUM_NODE_lon] = toDeg(lon1);
	rm[id] =  0;
}

__global__
void migrateGPU(double* children, int* rm, double* d, double* paramd, int* parami)
{

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id < parami[__DIMSUM_PARAMI_NUMCHILDREN])
		migrate(children, d, rm ,parami,paramd,id);


}
}
