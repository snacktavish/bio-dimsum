#include "hip/hip_runtime.h"
//  DIM SUM 2 -- Demography and Individual Migration Simulated Using a Markov chain
//  Copyright (C) 2011 Peter Hoffmann <p-hoffmann@web.de>
//
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "kernel.cuh"
#include <stdio.h>
#include <iostream>

#define DIMSUM_mode_northeast 0
#define DIMSUM_mode_southeast 1
#define DIMSUM_mode_northwest 2
#define DIMSUM_mode_southwest 3
#define PI 3.1415926536
#define DIMSUM_STD_R 6371.0f // radius of spherical earth in km (TODO: use value(# or units?) from xml)
#if __CUDA_ARCH__ < 130
#define DIMSUM_MIN_D 0.001
#define DIMSUM_KA2_EPSILON 0.00001745329252
#define DIMSUM_EPSILON 0.0017453292520000
#define EPSILON 0.00001
#else
#define DIMSUM_MIN_D 0.000001
#define DIMSUM_KA2_EPSILON 0.00000000001745329252
#define DIMSUM_EPSILON 0.00000017453292520000
#define EPSILON 0.000000000001
#endif

texture<float, 2, hipReadModeElementType> softborderDATA;
texture<float, 2, hipReadModeElementType> hardborderDATA;

hipArray* softborderArray;
hipArray* hardborderArray;

float *_sb_DATA;
float *_hb_DATA;
int _sbx, _sby, _hbx, _hby;

extern "C"
  {

    /**
     * Converts x from radians to decimal degree
     */
    __host__ __device__
    double toDeg(double x)
      { return x*180.0f/PI;}

    /**
     * Converts x from decimal degree to radians
     */
    __host__ __device__
    double toRad(double x)
      { return x*PI/180.0f;}

    /**
     * Modulus function for double
     */
    __host__ __device__
    double mod(double y, double x)
      {
        return (y - (x*floor(y/x)));
      }

    /**
     * Return 1 if lat >= sb_lat_bt and -1 if not
     */
    __host__ __device__
    double geq (double lat1, double sb_lat_bt)
      {
        if(lat1 >= sb_lat_bt)
        return 1;
        else //if (lat1 < sb_lat_bt)
        return -1;
      }

    /*
     * Generates a random number
     */
    __host__ __device__
    float nextRand(long* seed, int id)
      {
        seed[DIMSUM_PARAMI_RANDINDEX+id] = (seed[DIMSUM_PARAMI_RANDINDEX+id] * 0x5DEECE66DL + 0xBL) & ((1L << 48) - 1);
        float r = (int)(seed[DIMSUM_PARAMI_RANDINDEX+id] >> 24);
        r = r/((float) (1 << 24));
        //printf("%d %f\n", id, r);
        return r;
      }

    /**
     * Return the value of the softborder at (x,y)
     */
    __host__ __device__
    float sb_f(int x,int y)
      {
#ifdef __CUDA_ARCH__
        return tex2D(softborderDATA,x,y);
#else

        if (x < _sbx && y < _sby && x > -1 && y > -1)
          {
            return _sb_DATA[y * _sbx + x];
          }
        else
          {
            //std::cout << x << " " << y << std::endl;
            return 0;
          }
#endif
      }

    /**
     * Return the value of the hardborder at (x,y)
     */
    __host__ __device__
    float hb_f(int x,int y)
      {
#ifdef __CUDA_ARCH__
        return tex2D(hardborderDATA,x,y);
#else
        if (x < _hbx && y < _hby && x > -1 && y > -1)
          {
            return _hb_DATA[y * _hbx + x];
          }
        else
          {
            //std::cout << x << " " << y << std::endl;
            return 0;
          }
#endif
      }

    /**
     * Converts lng/lon to x/y coordinates of f
     */
    __host__ __device__
    int toX(double lon, double minlon, double maxlon, int img_size)
      {
        if( lon >= maxlon )
        return img_size -1;
        if( lon <= minlon )
        return 0;
        return (int)(img_size * (lon-minlon)/(maxlon-minlon));
      }

    /**
     * Returns the direction of the course
     */
    __host__ __device__
    int getMode(double crs)
      {
        int mode = -1;
        if( crs >= 0 && crs <= PI/2 )
          { // JMB COMMENT -- DETERMINES IF COURSE IS NORTHEAST...OR IS THIS SOUTHEAST, SINCE INPUT MAPS ARE FLIPPED??
            mode = DIMSUM_mode_northeast;
          }
        else if( crs > PI/2 && crs <= PI )
          { // JMB COMMENT -- Determines if course is southeast.  Note that this coordinate system is upside down relative to the standard.
            mode = DIMSUM_mode_southeast;
          }
        else if( crs > PI && crs <= 3*PI/2 )
          { // JMB COMMENT -- DETERMINES IF COURSE IS SOUTHWEST
            mode = DIMSUM_mode_southwest;
          }
        else if( crs > 3*PI/2 && crs <= 2*PI )
          { // JMB COMMENT -- 10.19.09 -- DETERMINES IF COURSE IS NORTHWEST
            mode = DIMSUM_mode_northwest;
          }
        return mode;
      }

    /**
     * Returns 1 if the direction is north and -1 if the direction is south
     */
    __host__ __device__
    int northORsouth(int i)
      {
        if(i == DIMSUM_mode_northeast || i == DIMSUM_mode_northwest)
        return 1;
        else return -1;
      }

    /**
     * Returns 1 if the direction is east and -1 if the direction is west
     */
    __host__ __device__
    int eastORwest(int i)
      {
        if(i == DIMSUM_mode_northeast || i == DIMSUM_mode_southeast)
        return 1;
        else return -1;
      }

    /**
     * Returns the next latspace on course
     */
    __host__ __device__
    double getI(double lat1, double minlat, double latspace, int dir)
      {
        double ilat;
        double mlat = lat1-minlat;
        if(dir == 1)
        ilat = minlat + (latspace*floor(mlat/latspace)) + latspace;
        else
        ilat = minlat + (latspace*floor(mlat/latspace));

        return ilat;
      }

    /**
     * JMB COMMENT -- 10.19.09 -- DETERMINES DISTANCE (D) FROM ENDING LAT/LON
     */
    __host__ __device__
    double dfromll(double lats, double lons, double latf, double lonf)
      {
        double dlat = latf - lats;
        double dlon = lonf - lons;
        double a = pow(sin(dlat/2),2) + cos(lats)*cos(latf)*pow(sin(dlon/2),2);

        return DIMSUM_STD_R * 2 * atan2(sqrt(a),sqrt(1-a));
      }

    /**
     * Determines distance to lon3 from lat1,lon1 over lat2,lon2
     */
    __host__ __device__
    double latdfromlon3(double lat1,double lon1, double lat2, double lon2, double lon3) // // Implementation by JMB from http://williams.best.vwh.net/avform.htm#Par  -- lat=atan(  (sin(lat1)*cosf(lat2)*sin(lon-lon2)-sin(lat2)*cosf(lat1)*sin(lon-lon1))  /  (cosf(lat1)*cosf(lat2)*sin(lon1-lon2))  )

      {
        double latf = atan(((sin(lat1)*cos(lat2)*sin(lon3-lon2))-(sin(lat2)*cos(lat1)*sin(lon3-lon1)))/(cos(lat1)*cos(lat2)*sin(lon1-lon2)));

        return dfromll(lat1,lon1,latf, lon3);
      }

    /**
     * Determines distance to lat3 from lat1,lon1 over lat2,lon2
     */
    __host__ __device__
    double londfromlat3(double lat1,double lon1, double lat2, double lon2, double lat3) // in degrees		JMB COMMENT -- 10.19.09 -- FINDS LONGITUDE AT WHICH NEAREST LAT IS CROSSED W/

      { //								GREAT CIRCLE DISTANCE
        // from http://williams.best.vwh.net/avform.htm#Par
        double d;

        double A = sin(lat1)*cos(lat2)*cos(lat3)*sin(lon1-lon2);
        double B = sin(lat1)*cos(lat2)*cos(lat3)*cos(lon1-lon2) - cos(lat1)*sin(lat2)*cos(lat3);
        double C = cos(lat1)*cos(lat2)*sin(lat3)*sin(lon1-lon2);
        double lon = atan2(B,A);
        if (abs(C) >sqrt(pow(A,2) + pow(B,2)))
          {
            d = 10000000;
          }
        else if( lat1 == lat3 )
          {
            d = 0;
          }
        else
          {
            double dlon = acos(C/sqrt(pow(A,2)+pow(B,2)));
            double lon3_1=(mod((lon1+dlon+lon+PI),(2*PI))-PI);
            double lon3_2=(mod((lon1-dlon+lon+PI),(2*PI))-PI);
            double lon3_1_d_D = dfromll(lat1, lon1,lat3,lon3_1);
            double lon3_2_d_D = dfromll(lat1, lon1,lat3, lon3_2);

            if( ((lon3_1 >= lon1 && lon3_1 <= lon2) || (lon3_1 <= lon1 && lon3_1 >= lon2)) && (lon3_1_d_D < lon3_2_d_D ))
            d = lon3_1_d_D;
            else
            d = lon3_2_d_D;

          }

        return d;
      }

    /**
     * Return X if X is not on x_bt and X + prefix*DIMSUM_EPSILON otherwise
     */
    __host__ __device__
    double setX(double X, double x_bt, double prefix, double dXY)
      {
        double r = X;
        if (((prefix > 0 && r < x_bt) || (prefix < 0 && r > x_bt)) && dXY == 0)
        r = x_bt + DIMSUM_EPSILON*prefix;
        return r;
      }

    /**
     * Returns the next latspace on course
     */
    __host__ __device__
    double nearestBorderCrossing(double lat1,double minlat, double sb_latspace, int dirLat,long* _index, int id)
      {
        double ilat = getI(lat1,minlat,sb_latspace, dirLat);
        if (abs(lat1-ilat) < EPSILON)//(lat1 == ilat)	// Checking to see if individual is starting RIGHT ON a lat line.

          { // If so, pushes it barely off in a random direction.
            if (nextRand(_index, id) > 0.5)
            lat1 += DIMSUM_KA2_EPSILON;
            else
            lat1 -= DIMSUM_KA2_EPSILON;
            ilat = getI(lat1,minlat,sb_latspace, dirLat);
          }
        return ilat;
      }

    __host__ __device__
    void migrate(double* node, double* dA, int* rm, long* parami, double* paramv, int id)
      {

        double d = dA[id];

        int sb_xsize = (int)parami[DIMSUM_PARAMI_SB_XSIZE];
        int sb_ysize = (int)parami[DIMSUM_PARAMI_SB_YSIZE];
        int hb_xsize = (int)parami[DIMSUM_PARAMI_HB_XSIZE];
        int hb_ysize = (int)parami[DIMSUM_PARAMI_HB_YSIZE];
        double lat1 = toRad(node[id*2+DIMSUM_NODE_lat]);
        double lon1 = toRad(node[id*2+DIMSUM_NODE_lon]);

        double minlat = toRad(paramv[0]);
        double maxlat = toRad(paramv[1]);
        double minlon = toRad(paramv[2]);
        double maxlon = toRad(paramv[3]);
        double sb_lonspace = toRad(paramv[4]);
        double sb_latspace = toRad(paramv[5]);
        double hb_lonspace = toRad(paramv[6]);
        double hb_latspace = toRad(paramv[7]);

        double step_d = 0.0001;
        double sb_lat_bt,sb_lon_bt,hb_lat_bt,hb_lon_bt,hb_dd,sb_dd; // JMB -- Using this to keep track of lat/lon value for border reflections and adjusting inexact positions, if necessary
        double crs = nextRand(parami,id) * 2 * PI; // Modified by JMB -- 4.5.10
        double lld_d, lld_latf=0, lld_lonf=0,lld_crs;

        while( d >= DIMSUM_MIN_D )
          {
            lld_d = d; // JMB COMMENT -- SETS D INTERNALLY IN LLD OBJECT
            lld_crs = crs; // JMB COMMENT -- SETS CRS INTERNALLY WITHIN LLD OBJECT
            lld_latf = asin(sin(lat1)*cos(lld_d/DIMSUM_STD_R)+cos(lat1)*sin(lld_d/DIMSUM_STD_R)*cos(lld_crs));// JMB COMMENT -- GETS ENDING LAT/LON FROM DISTANCE AND COURSE AND STORES INTERNALLY IN LLD OBJECT
            lld_lonf = lon1 + atan2(sin(lld_crs)*sin(lld_d/DIMSUM_STD_R)*cos(lat1),cos(lld_d/DIMSUM_STD_R)-sin(lat1)*sin(lld_latf));

            hb_dd=100000000;
            sb_dd=100000000;
            int hb_dx=0,hb_dy=0,sb_dx=0,sb_dy=0;

            int mode = getMode(crs);

            sb_lat_bt = nearestBorderCrossing(lat1,minlat,sb_latspace,northORsouth(mode),parami,id);
            double i1_d = londfromlat3(lat1,lon1,lld_latf,lld_lonf,sb_lat_bt); // JMB COMMENT -- FINDS COORDINATES FOR NEAREST LAT BORDER CROSSING

            sb_lon_bt = nearestBorderCrossing(lon1,minlon,sb_lonspace,eastORwest(mode),parami,id);
            double i2_d = latdfromlon3(lat1,lon1,lld_latf,lld_lonf,sb_lon_bt); // JMB COMMENT -- FINDS COORDINATES FOR NEAREST LON BORDER CROSSING

            if( i1_d <= i2_d && i1_d < d)
              {
                sb_dd = i1_d;
                sb_dy = northORsouth(mode);
              }
            else if( i2_d < d )
              {
                sb_dd = i2_d;
                sb_dx = eastORwest(mode);
              }

            hb_lat_bt = nearestBorderCrossing(lat1,minlat,hb_latspace,northORsouth(mode),parami,id);
            i1_d = londfromlat3(lat1,lon1,lld_latf,lld_lonf,hb_lat_bt); // JMB COMMENT -- FINDS COORDINATES FOR NEAREST LAT BORDER CROSSING

            hb_lon_bt = nearestBorderCrossing(lon1,minlon,hb_lonspace,eastORwest(mode),parami,id);
            i2_d = latdfromlon3(lat1,lon1,lld_latf,lld_lonf,hb_lon_bt); // JMB COMMENT -- FINDS COORDINATES FOR NEAREST LON BORDER CROSSING

            if( i1_d <= i2_d && i1_d < d)
              {
                hb_dd = i1_d;
                hb_dy = northORsouth(mode);
              }
            else if( i2_d < d )
              {
                hb_dd = i2_d;
                hb_dx = eastORwest(mode);
              }

            if( d < min(sb_dd,hb_dd) )
              { // JMB COMMENT -- 10.20.09 -- DISPERSAL OCCURS BEFORE PIXEL BOUNDARY IS CROSSED,
                lld_d = d;
                //*********************** Fudging to keep poorly estimated positions (due to step_d alterations) from crossing border boundaries inadvertently ***********************
                // IS THIS CHECK NECESSARY AT THIS POINT IN THE LOOP?  PERHAPS NOT, BUT SHOULD MAKE SURE BEFORE REMOVING IT.

                lat1 = setX(lld_latf,sb_lat_bt,geq(lat1,sb_lat_bt),0);
                lon1 = setX(lld_lonf,sb_lon_bt,geq(lon1,sb_lon_bt),0);
                lat1 = setX(lld_latf,hb_lat_bt,geq(lat1,hb_lat_bt),0);
                lon1 = setX(lld_lonf,hb_lon_bt,geq(lon1,hb_lon_bt),0);
                d=0;
              }
            else if( abs(sb_dd-hb_dd) < step_d || sb_dd < hb_dd )
              { // JMB COMMENT -- 10.20.09 -- BOTH SOFT AND HARD PIXEL BOUNDARIES WILL BE CROSSED
                // both soft & hard must be checked at the same time -- but the order is up to you
                // I arbitrarily chose to check hard borders first
                if(abs(sb_dd-hb_dd) < step_d )
                if( nextRand(parami,id) <= hb_f(toX(lon1,minlon,maxlon,hb_xsize)+hb_dx,toX(lat1,minlat,maxlat,hb_ysize)+hb_dy) )
                  { // JMB COMMENT -- FINDS HARD BORDER VALUE FOR NEXT PIXEL WITH RESPECT TO LONGITUDE AND CHECKS TO SEE IF INDIVIDUAL SURVIVES HARD BORDER CROSSING.
                    rm[id] = 1; //continue childrenloop; // this exits the travel loop immediately, so the current child never gets added to the next generation
                    return;
                  }

                if( nextRand(parami,id) <= sb_f(toX(lon1,minlon,maxlon,sb_xsize)+sb_dx, toX(lat1,minlat,maxlat,sb_ysize)+sb_dy) )
                  {
                    // failed the soft border-- stop before border, reflect back, update d, and continue
                    crs = nextRand(parami,id) * 2 * PI;// / 4+3*PI/4;
                    sb_dy = 0;
                    sb_dx = 0;
                    lld_d = (sb_dd-step_d);
                  }
                else
                  {
                    lld_d = sb_dd+step_d;
                  }

                d-= lld_d;
                lld_latf = asin(sin(lat1)*cos(lld_d/DIMSUM_STD_R)+cos(lat1)*sin(lld_d/DIMSUM_STD_R)*cos(lld_crs));
                lld_lonf = lon1 + atan2(sin(lld_crs)*sin(lld_d/DIMSUM_STD_R)*cos(lat1),cos(lld_d/DIMSUM_STD_R)-sin(lat1)*sin(lld_latf));
                // Fudging to keep poorly estimated positions from crossing boundaries
                lat1 = setX(lld_latf,sb_lat_bt,geq(lat1,sb_lat_bt),sb_dy);
                lon1 = setX(lld_lonf,sb_lon_bt,geq(lon1,sb_lon_bt),sb_dx);
              }
            else
              {
                if( nextRand(parami,id) <= hb_f(toX(lon1,minlon,maxlon,hb_xsize)+hb_dx,toX(lat1,minlat,maxlat,hb_ysize)+hb_dy) )
                  {
                    rm[id] = 1; //continue childrenloop; // this exits the travel loop immediately, so the current child never gets added to the next generation
                    return; // JMB -- Would this lead to pruning problems?
                  }
                else
                  {
                    lld_d = hb_dd+step_d;
                    d-=lld_d;
                    lat1 = setX(lld_latf,hb_lat_bt,geq(lat1,hb_lat_bt),hb_dy);
                    lon1 = setX(lld_lonf,hb_lon_bt,geq(lon1,hb_lon_bt),hb_dx);
                  }
              }
          }
        node[id*2+DIMSUM_NODE_lat] = toDeg(lat1);
        node[id*2+DIMSUM_NODE_lon] = toDeg(lon1);
        //	printf("node: %lf %lf\n", node[id*2+DIMSUM_NODE_lat],node[id*2+DIMSUM_NODE_lon]);
        rm[id] = 0;
      }

    __global__
    void migrateGLOBAL(double* children, int* rm, double* d, double* paramd, long* parami, int size)
      {
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        /*if(id == 0) {
         for(int i=0;i<size+8;i++)
         printf("%d ",parami[i]);
         printf("\n");
         }*/
        if(id < size)
          {
            //printf("test %d %d\n",id, size);
            migrate(children, d, rm ,parami,paramd,id);
          }

      }

    void setArraysCPU(const float* sb_DATA, int sbx, int sby, const float* hb_DATA, int hbx, int hby)
      {
        _sbx = sbx;
        _sby = sby;
        _hbx = hbx;
        _hby = hby;
        _sb_DATA = new float[sbx*sby];
        _hb_DATA = new float[hbx*hby];
        for(int i=0;i<sbx*sby;i++)
        _sb_DATA[i] = sb_DATA[i];
        for(int i=0;i<hbx*hby;i++)
        _hb_DATA[i] = hb_DATA[i];
      }

    hipArray* cp2Texture(const float* sb_DATA, int sbx, int sby, texture<float, 2, hipReadModeElementType>& tex)
      {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipArray* cu_array;
        hipMallocArray( &cu_array, &channelDesc, sbx, sby );

        hipMemcpy2DToArray( cu_array, 0,0, sb_DATA, sbx*sizeof(float), sbx*sizeof(float), sby, hipMemcpyHostToDevice);

        tex.addressMode[0] = hipAddressModeClamp;
        tex.addressMode[1] = hipAddressModeClamp;
        tex.filterMode = hipFilterModePoint;
        tex.normalized = false;

        hipBindTextureToArray( tex, cu_array, channelDesc);
        return cu_array;
      }

    void setArraysGPU(const float* sb_DATA, int sbx, int sby, const float* hb_DATA, int hbx, int hby)
      {
        hipFreeArray(softborderArray);
        softborderArray = cp2Texture(sb_DATA,sbx,sby,softborderDATA);
        hipFreeArray(hardborderArray);
        hardborderArray = cp2Texture(hb_DATA,hbx,hby,hardborderDATA);
      }

    void migrateGPU(double* children, int* rm, double* d, double* paramd, long* parami)
      {

        int size = parami[DIMSUM_PARAMI_NUMCHILDREN];
        int block_size =128;
        int block_num = (int)ceil((double)size/(double)block_size);

        //CHILDREN
        double* d_children = NULL;
        hipMalloc( (void**) &d_children, sizeof(double)*size*2);
        hipMemcpy( d_children,children, sizeof(double)*size*2, hipMemcpyHostToDevice);

        //RM
        int* d_rm = NULL;
        hipMalloc( (void**) &d_rm, sizeof(int)*size);
        hipMemcpy(d_rm, rm, sizeof(int)*size, hipMemcpyHostToDevice);

        //d
        double* d_d = NULL;
        hipMalloc( (void**) &d_d, sizeof(double)*size);
        hipMemcpy(d_d, d, sizeof(double)*size, hipMemcpyHostToDevice);

        //paramd
        double* d_paramd = NULL;
        hipMalloc( (void**) &d_paramd, sizeof(double)*8);
        hipMemcpy(d_paramd, d, sizeof(double)*8, hipMemcpyHostToDevice);

        //parami
        long* d_parami = NULL;
        hipMalloc( (void**) &d_parami, sizeof(long)*(size+DIMSUM_PARAMI_RANDINDEX));
        hipMemcpy( d_parami, parami, sizeof(long)*(size+DIMSUM_PARAMI_RANDINDEX), hipMemcpyHostToDevice);

        migrateGLOBAL<<< block_num, block_size>>>( d_children, d_rm, d_d, d_paramd,d_parami, size);
        //hipCtxSynchronize();

#if CUDART_VERSION >= 4000
        hipDeviceSynchronize();
#else
        hipDeviceSynchronize();
#endif

        hipMemcpy( children,d_children, sizeof(double)*size*2, hipMemcpyDeviceToHost);
        hipMemcpy( rm,d_rm, sizeof(int)*size, hipMemcpyDeviceToHost);

        hipFree(d_parami);
        hipFree(d_paramd);
        hipFree(d_d);
        hipFree(d_rm);
        hipFree(d_children);

      }

    void migrateCPU(double* children, int* rm, double* d, double* paramd, long* parami)
      {
        for(int id =0;id < parami[DIMSUM_PARAMI_NUMCHILDREN];id++)
          {
            migrate(children, d, rm ,parami,paramd,id);
          }

      }

    void initGPU(const float* sb_DATA, int sbx, int sby, const float* hb_DATA, int hbx, int hby)
      {
        int devID;
        hipDeviceProp_t props;

        hipGetDevice(&devID);
        hipGetDeviceProperties(&props, devID);
        printf("Device %d: \"%s\" with Compute %d.%d capability\n\n",
            devID, props.name, props.major, props.minor);

        softborderArray = cp2Texture(sb_DATA,sbx,sby,softborderDATA);
        hardborderArray = cp2Texture(hb_DATA,hbx,hby,hardborderDATA);
      }

    void shutdownGPU()
      {
        hipFreeArray(softborderArray);
        hipFreeArray(hardborderArray);
      }
  }

